#include "hip/hip_runtime.h"
__global__ void calcNeurons(
        float t, unsigned layer_size,
        unsigned *spike_count, unsigned *spikes,
        float *spike_time, bool *fired)
{
    unsigned id = BLOCK_SIZE * blockIdx.x + threadIdx.x;

    if (id < layer_size) {
        __shared__ unsigned spikes_block[BLOCK_SIZE];
        __shared__ volatile unsigned spike_count_block;
        __shared__ volatile unsigned spikes_idx;

        if (threadIdx.x == 0) {
            spike_count_block = 0;
        }
        __syncthreads();

        ////////////////////////////////////////////////////////////
        // begin
        ////////////////////////////////////////////////////////////
        if (t >= spike_time[id] && !fired[id]) {
            fired[id] = true;
            spikes_block[atomicAdd((unsigned *)&spike_count_block, 1)] = id;
        }
        ////////////////////////////////////////////////////////////
        // end
        ////////////////////////////////////////////////////////////

        __syncthreads();
        if (threadIdx.x == 0) {
            if (spike_count_block > 0) {
                spikes_idx = atomicAdd(&spike_count[0], spike_count_block);
            }
        }

        __syncthreads();
        if (threadIdx.x < spike_count_block) {
            spikes[spikes_idx + threadIdx.x] = spikes_block[threadIdx.x];
        }
    }
}
